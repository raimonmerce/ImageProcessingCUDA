#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

__global__ void horizontal(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols, float *d_vec, int ksize)
{
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  const int row = blockDim.y * blockIdx.y + threadIdx.y;

  int sizeDiv2 = ksize/2;
  //MAX KSIZE = 15
  __shared__ uchar3 temp[32][32 + 15];

  int xindex = threadIdx.x + sizeDiv2;
  int yindex = threadIdx.y + sizeDiv2;
  if (col > ksize && col < cols - 32) {
    if (threadIdx.y < sizeDiv2) {
      temp[yindex][xindex - sizeDiv2] = src(row, col  - sizeDiv2);
      temp[yindex][xindex + 32] = src(row, col + 32);
    }
  }
  __syncthreads();

  int startM_r = row - sizeDiv2;
  int startM_c = col - sizeDiv2;

  float total = 0.0;
  float r = 0.0;
  float g = 0.0;
  float b = 0.0;

  for (int i = 0; i < ksize; i++) {
    if ((startM_r + i) >= 0 && (startM_r + i) < rows) {
      uchar3 val;
      if (threadIdx.x < sizeDiv2 && threadIdx.x > 32 - sizeDiv2){
        val = temp[threadIdx.y][threadIdx.x + i];   
      } else val = src(startM_r, startM_c + i);
      r += float(val.x) * d_vec[i];
      g += float(val.y) * d_vec[i];
      b += float(val.z) * d_vec[i];
      total += d_vec[i];
    }
  }
  unsigned char rf = r/total;
  unsigned char gf = g/total;
  unsigned char bf = b/total;

  dst(row, col).x = rf;
  dst(row, col).y = gf;
  dst(row, col).z = bf;
}

__global__ void vertical(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols, float *d_vec, int ksize)
{
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  const int row = blockDim.y * blockIdx.y + threadIdx.y;

  int sizeDiv2 = ksize/2;
  //MAX KSIZE = 15
  __shared__ uchar3 temp[32 + 15][32];

  int xindex = threadIdx.x + sizeDiv2;
  int yindex = threadIdx.y + sizeDiv2;
  if (row > ksize && row < rows - 32) {
    if (threadIdx.y < sizeDiv2) {
      temp[yindex - sizeDiv2][xindex] = src(row - sizeDiv2, col );
      temp[yindex + 32][xindex] = src(row + 32, col);
    }
  }
  __syncthreads();

  int startM_r = row - (ksize/2);
  int startM_c = col - (ksize/2);

  float total = 0.0;
  float r = 0.0;
  float g = 0.0;
  float b = 0.0;

  for (int j = 0; j < ksize; j++) {
    if ((startM_c + j) >= 0 && (startM_c + j) < cols) {
            uchar3 val;
      if (threadIdx.y < sizeDiv2 && threadIdx.y > 32 -sizeDiv2){
        val = temp[threadIdx.y + j][threadIdx.x];   
      } else val = src(startM_r + j, startM_c);
      r += float(val.x) * d_vec[j];
      g += float(val.y) * d_vec[j];
      b += float(val.z) * d_vec[j];
      total += d_vec[j];
    }
  }

  unsigned char rf = r/total;
  unsigned char gf = g/total;
  unsigned char bf = b/total;

  dst(row, col).x = rf;
  dst(row, col).y = gf;
  dst(row, col).z = bf;
}

int divUp(int a, int b)
{
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA ( cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, cv::cuda::GpuMat& tmp, float *d_vec, int ksize)
//void startCUDA ( cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, float *d_vec, int ksize)
{
  int THREADS = 32;
  int BLOCKS = (dst.cols + THREADS - 1) / THREADS;
  dim3 block_dim(THREADS, THREADS);
  dim3 grid_dim(BLOCKS, BLOCKS);
  horizontal<<<grid_dim, block_dim>>>(src, tmp, dst.rows, dst.cols, d_vec, ksize);
  vertical<<<grid_dim, block_dim>>>(tmp, dst, dst.rows, dst.cols, d_vec, ksize);
}

