#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <opencv2/opencv.hpp>
#include <cfloat>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/core/cuda/border_interpolate.hpp>
#include <opencv2/core/cuda/vec_traits.hpp>
#include <opencv2/core/cuda/vec_math.hpp>

__device__ bool smaller(uchar3 a, uchar3 b) {
  float fa = float(a.x) + float(a.y) + float(a.z);
  float fb = float(b.x) + float(b.y) + float(b.z);
  return fa < fb;
}

__global__ void process(const cv::cuda::PtrStep<uchar3> src, cv::cuda::PtrStep<uchar3> dst, int rows, int cols, int ksize, int percent )
{
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  const int row = blockDim.y * blockIdx.y + threadIdx.y;

  int sizeDiv2 = ksize/2;
  __shared__ uchar3 temp[32 + 15][32 + 15];
  int xindex = threadIdx.x + sizeDiv2;
  int yindex = threadIdx.y + sizeDiv2;
  if ((col > ksize && col < cols - 32) && (row > ksize && row < rows - 32)) {
    
    if (threadIdx.x < sizeDiv2 && threadIdx.y < sizeDiv2) {
      temp[yindex - sizeDiv2][xindex - sizeDiv2]  = src(row - sizeDiv2, col - sizeDiv2);
      temp[yindex + 32][xindex - sizeDiv2]  = src(row + 32, col - sizeDiv2);
      temp[yindex - sizeDiv2][xindex + 32] = src(row - sizeDiv2, col + 32);
      temp[yindex + 32][xindex + 32] = src(row + 32, col + 32);
    }

    if (threadIdx.x < sizeDiv2) {
      temp[yindex] [xindex - sizeDiv2] = src(row, col - sizeDiv2);
      temp[yindex] [xindex + 32] = src(row, col + 32);
    }

    if (threadIdx.y < sizeDiv2) {
      temp[yindex - sizeDiv2][xindex] = src(row- sizeDiv2, col );
      temp[yindex + 32][xindex] = src(row + 32, col);
    }
    

    //temp[yindex][xindex] = src(row, col);
  }
  __syncthreads();
  int startM_r = row - sizeDiv2;
  int startM_c = col - sizeDiv2;

  //uchar3 val = src(dst_y, dst_x);
  float total = 0.0;
  int val = (ksize-1)/2;
  int count = 0;
  uchar3 sorted[255];

  for (int i = 0; i < ksize; i++) {
    for (int j = 0; j < ksize; j++) {
      if ((startM_r + i) >= 0 && (startM_r + i) < rows) {
        if ((startM_c + j) >= 0 && (startM_c + j) < cols) {
          uchar3 key;
          if (threadIdx.x < sizeDiv2 && threadIdx.y < sizeDiv2
            && threadIdx.x > 32 - sizeDiv2 && threadIdx.y > 32 -sizeDiv2)
            key = temp[threadIdx.y + j][threadIdx.x + i]; 
          else key = src(startM_r + j, startM_c + i);
          int k = count - 1;
          while (k >= 0 && smaller(key, sorted[k])){
              sorted[k + 1] = sorted[k];
              --k;
          }
          sorted[k + 1] = key;
          count++;
        }
      }
    }
  }

  int hallf = int(((int((ksize*ksize) / 2) - 1) * percent) / 100);
  int ini = int((ksize*ksize) / 2) - hallf;
  int end = ((ksize*ksize) / 2) + hallf;
  if ((ksize*ksize) % 2 == 0) ++end;

  float b = 0.0;
  float g = 0.0;
  float r = 0.0;

  for (int k = ini; k <= end; k++) {
      r += float(sorted[k].x);
      g += float(sorted[k].y);
      b += float(sorted[k].z);
  }
  float div = float(end - ini + 1);

  unsigned char rf = r/div;
  unsigned char gf = g/div;
  unsigned char bf = b/div;

  dst(row, col).x = rf;
  dst(row, col).y = gf;
  dst(row, col).z = bf;
}

int divUp(int a, int b)
{
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void startCUDA (cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, int ksize, int percent)
{
  int THREADS = 32;
  int BLOCKS = (dst.cols + THREADS - 1) / THREADS;
  dim3 block_dim(THREADS, THREADS);
  dim3 grid_dim(BLOCKS, BLOCKS);

  process<<<grid_dim, block_dim>>>(src, dst, dst.rows, dst.cols, ksize, percent);

}

